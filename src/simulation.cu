//
// Created by goader on 11/5/23.
//

#include "simulation.cuh"


Simulation::Simulation(const SimulationConfig& config) : config(config), rng(config) {
    h_particles = new Particle[config.numParticles];

    d_allFrozen = nullptr;
    d_particles = nullptr;
    d_states = nullptr;

    d_forceFieldX = nullptr;
    d_forceFieldY = nullptr;
    d_obstacles = nullptr;

    numBlocks1d = (config.numParticles + BLOCK_SIZE_1D - 1) / BLOCK_SIZE_1D;
    numBlocks2d = (config.numParticles + BLOCK_SIZE_2D - 1) / BLOCK_SIZE_2D;
}

Simulation::~Simulation() {
    delete[] h_particles;
    hipFree(d_particles);
    hipFree(d_states);
    hipFree(d_allFrozen);
    hipFree(d_forceFieldX);
    hipFree(d_forceFieldY);
    hipFree(d_obstacles);
}

void Simulation::initParticles(std::vector<Particle> initialParticles) {
    size_t frozenParticles = initialParticles.size();

    for (int i = 0; i < frozenParticles; i++) {
        h_particles[i].oldX = initialParticles[i].oldX;
        h_particles[i].oldY = initialParticles[i].oldY;
        h_particles[i].x = initialParticles[i].x;
        h_particles[i].y = initialParticles[i].y;
        h_particles[i].isActive = initialParticles[i].isActive;
        h_particles[i].frozenAtStep = initialParticles[i].frozenAtStep;
        h_particles[i].collidedParticleIdx = initialParticles[i].collidedParticleIdx;
    }

    for (size_t i = frozenParticles; i < config.numParticles; i++) {
        auto x = rng.generateParticleX();
        auto y = rng.generateParticleY();
        h_particles[i].oldX = x;
        h_particles[i].oldY = y;
        h_particles[i].x = x;
        h_particles[i].y = y;
        h_particles[i].isActive = true;
        h_particles[i].frozenAtStep = -100;
        h_particles[i].collidedParticleIdx = -1;
    }
}

void Simulation::setupCudaForceField(float* forceFieldX, float* forceFieldY) {
        hipMalloc(&d_forceFieldX, config.width * config.height * sizeof(float));
        hipMemcpy(d_forceFieldX, forceFieldX, config.width * config.height * sizeof(float), hipMemcpyHostToDevice);

        hipMalloc(&d_forceFieldY, config.width * config.height * sizeof(float));
        hipMemcpy(d_forceFieldY, forceFieldY, config.width * config.height * sizeof(float), hipMemcpyHostToDevice);
}


void Simulation::setupCudaObstacles(std::vector<Obstacle> obstacles) {
    size_t numObstacles = obstacles.size();
    
    auto h_obstacles = new Obstacle[numObstacles];
    for (int i = 0; i < numObstacles; i++) {
        h_obstacles[i].xTopLeft = obstacles[i].xTopLeft;
        h_obstacles[i].yTopLeft = obstacles[i].yTopLeft;
        h_obstacles[i].recHeight = obstacles[i].recHeight;
        h_obstacles[i].recWidth = obstacles[i].recWidth;
    }

    hipError_t err;

    err = hipMalloc(&d_obstacles, numObstacles * sizeof(Obstacle));
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        delete[] h_obstacles;
        return;
    }
    
    err = hipMemcpy(d_obstacles, h_obstacles, numObstacles * sizeof(Obstacle), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        hipFree(d_obstacles);
        delete[] h_obstacles;
        return;
    }

    delete[] h_obstacles;
}


// must be called after initParticles
void Simulation::setupCuda() {
    hipMalloc(&d_allFrozen, sizeof(bool));

    hipMalloc(&d_states, config.numParticles * sizeof(hiprandState));
    setupRandomStatesKernel<<<numBlocks1d, BLOCK_SIZE_1D>>>(d_states, config.seed);

//    hipDeviceSynchronize();
//    hipError_t error = hipGetLastError();
//    if(error != hipSuccess)
//    {
//        // print the CUDA error message and exit
//        printf("CUDA error: %s\n", hipGetErrorString(error));
//        exit(-1);
//    }
//    else {
//        printf("Success!\n");
//    }

    hipMalloc(&d_particles, config.numParticles * sizeof(Particle));
    hipMemcpy(d_particles, h_particles,
               config.numParticles * sizeof(Particle), hipMemcpyHostToDevice);
}

void Simulation::step() {
    moveParticlesKernel<<<numBlocks1d, BLOCK_SIZE_1D>>>(
            d_particles,
            config,
            d_states,
            d_forceFieldX,
            d_forceFieldY,
            d_obstacles
    );

    hipMemset(d_allFrozen, 1, sizeof(bool));  // set d_allFrozen to true
    dim3 gridDims(numBlocks2d, numBlocks2d); dim3 blockDims(BLOCK_SIZE_2D, BLOCK_SIZE_2D);
    checkCollisionsKernel<<<gridDims, blockDims>>>(
            d_particles,
            config
    );

    freezeParticlesKernel<<<numBlocks1d, BLOCK_SIZE_1D>>>(
            d_particles,
            config,
            d_states,
            d_allFrozen,
            current_step
    );
    hipDeviceSynchronize();  // waiting for the d_allFrozen to be updated
    hipMemcpy(&h_allFrozen, d_allFrozen, sizeof(bool), hipMemcpyDeviceToHost);

    current_step++;
}

std::vector<Particle> Simulation::getParticles() {
    hipDeviceSynchronize();
    hipMemcpy(h_particles, d_particles,
               config.numParticles * sizeof(Particle), hipMemcpyDeviceToHost);

    // copy the particles to a vector
    std::vector<Particle> particles;
    particles.reserve(config.numParticles);
    for (int i = 0; i < config.numParticles; i++) {
        particles.push_back(h_particles[i]);
    }
    return particles;
}

int Simulation::getCurrentStep() const {
    return current_step;
}

bool Simulation::isFinished() const {
    return h_allFrozen;
}
